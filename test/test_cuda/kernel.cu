#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <cuda_tool.h>


__global__ void Add(int *a, int *b, int *c, int DX)
{
    int f = blockIdx.x*blockDim.x + threadIdx.x;

    if (f >= DX) return;

    c[f] = a[f] + b[f];

}

void AddKernel(int *a, int *b, int *c, int DX)
{
    dim3 dimBlock = (32);
    dim3 dimGrid = ((DX + 32 - 1) / 128);
    Add << <dimGrid, dimBlock >> > (a, b, c, DX);
    hipDeviceSynchronize();
}